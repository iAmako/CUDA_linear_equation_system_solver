#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>

double wtime(void)
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + tv.tv_usec * 1e-6;
}


__global__ void solve(double** system, int* len){

}

int main(void){
    if(argc < 2){
        printf("Utilisation : ./solver_cuda.exe PATH\nAvec :\n\tPATH : Chemin vers le fichier\n");
        return EXIT_SUCCESS;
    }

    for(int i= 0 ; i < 5 ; i++){
        n[i]= i ;
    }

    hipMalloc((void **));

    hipMemcpy(d_n, n, sizeof(), hipMemcpyHostToDevice);

    solve<<<1,1>>>(d_n);

    hipMemcpy( n, d_n, sizeof(), hipMemcpyDeviceToHost);
    
    free(n);
    hipFree(d_n);
    return 0;   
}